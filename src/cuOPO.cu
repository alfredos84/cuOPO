#include "hip/hip_runtime.h"
/* Author Alfredo Daniel Sanchez: alfredo.daniel.sanchez@gmail.com */

// Necessary headers
#include <iostream>
#include <string>
#include <random>
#include <fstream>
#include <iomanip>
#include <typeinfo>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <stdio.h>
#include <chrono>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

// Single precision Real and complex data types
using real_t = float;
using complex_t = hipfftComplex;

// Define global constants
const real_t PI   = 3.14159265358979323846;	// pi
const real_t C    = 299792458*1E6/1E12;		// speed of ligth in vacuum [um/ps]
const real_t EPS0 = 8.8541878128E-12*1E12/1E6;	// vacuum pertivity [W.ps/V²μm] 

const uint SIZE   = 1 << 14;				// vector size
const uint NZ     = 150;				// steps over z direction
const uint NRT    = 10000;				// number of round trips            
const uint BLKX   = 16;					// block dimensions for kernels

// Package headers
#include "headers/common.h"
#include "headers/operators.h"
#ifdef PPLN // Mgo:PPLN nonlinear crystal
#include "headers/ppln.h"
#endif
#ifdef SPPLT // Mgo:sPPLT nonlinear crystal
#include "headers/spplt.h"
#endif
#include "headers/functions.h"
#ifdef THREE_EQS // Define 2 or 3 coupled-wave equations
#include "headers/cwes3.h"
#else
#include "headers/cwes2.h"
#endif
#include "headers/files.h"



int main(int argc, char *argv[]){
	
	std::cout << "\n\n\n#######---Welcome to OPO simulator---#######\n\n\n" << std::endl;
	
	////////////////////////////////////////////////////////////////////////////////////////
	//* Setting GPU and timing */
	
	time_t current_time; // timing the code
	time(&current_time);
	std::cout << "Date: " << ctime(&current_time) << std::endl;
	double iStart = seconds();
	
	
	#ifdef CW_OPO
	std::cout << "Regime: continuous wave" << std::endl;
	#endif
	#ifdef NS_OPO
	std::cout << "Regime: nanosecond" << std::endl;
	#endif
	#ifdef THREE_EQS
	std::cout << "Three equations" << std::endl;
	#else
	std::cout << "Two equations" << std::endl;
	#endif
	
	// Set up device (GPU)
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	std::cout << "\n\nUsing Device " << dev << ": GPU " << deviceProp.name << std::endl;
	CHECK(hipSetDevice(dev));
	////////////////////////////////////////////////////////////////////////////////////////
	
	
	
	
	////////////////////////////////////////////////////////////////////////////////////////
	//* Define simulation parameters, physical quantities and set electric fields */
	
	// Grids, crystal and cavity parameters //
	real_t lp        = atof(argv[1])*1e-3;  // pump wavelength   [μm]
	real_t ls        = 2*lp;                // signal wavelength [μm]
	real_t li        = lp*ls/(ls-lp);       // idler wavelength  [μm]
	
	real_t Temp      = atof(argv[2]);       // crystal temperature [ºC]
	real_t Lambda    = atof(argv[3]);       // grating period for QPM [μm]  
	real_t Lcr       = 5e3;                 // crystal length [um]
	
	real_t np        = n(lp, Temp);         // pump ref. index
	real_t vp        = group_vel(lp, Temp); // pump group velocity [μm/ps]
	real_t b2p       = gvd(lp, Temp);       // pump GVD [ps²/μm] 
	real_t b3p       = 0.*TOD(lp, Temp);    // pump TOD [ps³/μm]	
	real_t kp        = 2*PI*deff/(np*lp);   // pump kappa [1/V]
	
	real_t ns        = n(ls, Temp);         // signal ref. index
	real_t vs        = group_vel(ls, Temp); // signal group velocity [μm/ps]
	real_t b2s       = gvd(ls, Temp);       // signal GVD [ps²/μm] 
	real_t b3s       = 0.*TOD(ls, Temp);    // signal TOD [ps³/μm]		
	real_t ks        = 2*PI*deff/(ns*ls);   // signal kappa [1/V]
	
	real_t ni        = n(li, Temp);         // idler ref. index
	real_t vi        = group_vel(li, Temp); // idler group velocity [μm/ps]
	real_t b2i       = gvd(li, Temp);       // idler GVD [ps²/μm]
	real_t b3i       = 0.*TOD(li, Temp);    // idler TOD [ps³/μm]	
	real_t ki        = 2*PI*deff/(ni*li);   // idler kappa [1/V]
	
	real_t dk        = 2*PI*( np/lp-ns/ls-ni/li-1/Lambda ); // mismatch factor
	real_t dkp       = 1/vp-1/vs;                           // group velocity mismatch	
	real_t Lcav      = 5 * Lcr;                             // cavity length [um]
	real_t Rs        = atof(argv[5])*0.01;                  // Reflectivity at signal wavelength 
	real_t alphas    = 0.5*((1-Rs)+alpha_crs*Lcr);          // Total losses for threshold condition signal
	#ifdef THREE_EQS
	real_t Ri        = 0.98;                                // Reflectivity at idler wavelength 
	real_t alphai    = 0.5*((1-Ri)+alpha_cri*Lcr);          // Total losses for threshold condition idler
	#endif
	real_t t_rt      = (Lcav+Lcr*(ns-1))/C;                 // round-trip time [ps]
	real_t FSR       = 1/t_rt;	                          // free-spectral range
	real_t finesse   = 2*PI/(1-Rs);                         // cavity finesse
	real_t lw        = FSR/finesse*1e6;                     // cavity Linewidth [MHz]
	real_t delta     = atof(argv[6]);                       // cavity detuning [rad] 
	real_t epsilon   = atof(argv[7])*0.01;                  // dispersion compensation index
	real_t GDD       = -epsilon*b2s*Lcr;                    // GDD [ps²]
	real_t TODscomp  = -0.01*atof(argv[8])*b3s*Lcr;         // TOD compensation [ps³]
	real_t TODicomp  = -0.01*atof(argv[8])*b3i*Lcr;         // TOD compensation [ps³]
	
	
	// z discretization, time and frequency discretization
	real_t dz        = Lcr/NZ;    // number of z-steps in the crystal
	real_t dT        = t_rt/SIZE; // time step in [ps]
	real_t dF        = 1/t_rt;    // frequency step in [THz]
	
	
	bool stride       = false;
	uint Nrts;        // number of last round trips to save (only for cw)
	if(stride){Nrts = 100;}
	else{Nrts = 16;}
	
	
	#ifdef CW_OPO
	uint SIZEL = SIZE*Nrts;     // size of large vectors for full simulation
	real_t T_width = t_rt*Nrts; // total time for the saved simulation
	#endif
	#ifdef NS_OPO
	uint SIZEL = SIZE*NRT; // size of large vectors for full simulation
	real_t T_width = t_rt*NRT; // total time for the saved simulation
	#endif	
	
	// Time vector T for one round trip
	real_t *T = (real_t*) malloc(sizeof(real_t) * SIZE);
	linspace( T, SIZE, -0.5*t_rt, 0.5*t_rt);
	
	// Time vector Tp for full simulation
	real_t *Tp = (real_t*) malloc(sizeof(real_t) * SIZEL);
	linspace( Tp, SIZEL, -0.5*T_width, 0.5*T_width);
	
	// Time vector Fp for full simulation
	real_t dFp  = 1/T_width;
	real_t *Fp = (real_t*) malloc(sizeof(real_t) * SIZEL);
	linspace( Fp, SIZEL, -0.5*SIZEL*dFp, +0.5*SIZEL*dFp);
	
	// Frequency and angular frequency vectors f and Ω
	real_t *F = (real_t*) malloc(sizeof(real_t) * SIZE);
	linspace( F, SIZE, -0.5*SIZE*dF, +0.5*SIZE*dF);
	real_t *w = (real_t*) malloc(sizeof(real_t) * SIZE);
	fftshift(w,F, SIZE);
	for (uint i=0; i<SIZE; i++){
		w[i] = 2*PI*w[i]; // angular frequency [2*pi*THz]
	}
	
	// Define memory size for complex host vectors
	uint nBytes   = sizeof(complex_t)*SIZE;
	
	// Difine which fields are resonant (SRO, DRO or TRO)
	bool is_Ap_resonant = false;
	bool is_As_resonant = true;
	#ifdef THREE_EQS
	bool is_Ai_resonant = true;
	#endif
	
	// Define input pump parameters
	real_t waist = 55;             // beam waist radius [um]
	real_t spot  = PI*waist*waist; // spot area [μm²]
	real_t Ith, Pth;               // Power and intensity threshold 
	#ifdef THREE_EQS
	// Power and intensity threshold non-degenerate OPO 
	if (!is_Ai_resonant){
		std::cout << "SRO: As is resonant" << std::endl;
		Ith   = EPS0*C*np*ns*ni*ls*li*pow((1/deff/Lcr/PI),2)*alphas/2;
	}
	if (!is_As_resonant){
		std::cout << "SRO, Ai is resonant" << std::endl;
		Ith   = EPS0*C*np*ns*ni*ls*li*pow((1/deff/Lcr/PI),2)*alphai/2;
	}
	if (is_As_resonant and is_Ai_resonant){
		std::cout << "DRO, As and Ai are resonant" << std::endl;
		Ith   = EPS0*C*np*ns*ni*ls*li*pow((1/deff/Lcr/PI),2)*alphas*alphai/8;
	}	
	Pth   = Ith*spot;
	#else
	// Power and intensity threshold degenerate DRO 
	Ith   = EPS0*C*np*powf((ns*ls*alphas/deff/Lcr/PI), 2)/8;
	Pth   = Ith*spot;
	#endif
	
	real_t Nth   = atof(argv[4]);             // Times over the threshold
	real_t Inten = atof(argv[4])*Ith;         // Pump intensity in [W/um²]
	real_t Power = Inten*spot;                // Pump power in [W]
	real_t Ap0   = sqrt(2*Inten/(np*EPS0*C)); // Input pump field strength [V/μm]
	
	// Define input pump vector
	#ifdef CW_OPO
	complex_t *Ap_in = (complex_t*)malloc(nBytes); // input pump vector
	input_field_T(Ap_in, Ap0, SIZE );              // set input pump vector (cw)
	#endif
	
	#ifdef NS_OPO
	real_t FWHM      = 10000;                              // intensity FWHM for input [ps]
	real_t sigmap    = FWHM*sqrtf(2)/(2*sqrtf(2*logf(2))); // σ of electric field gaussian pulse [ps]
	complex_t *Ap_in = (complex_t*)malloc(sizeof(complex_t)*SIZEL); // input pump vector
	input_field_T(Ap_in, Ap0, Tp, sigmap, SIZEL); // set input pump vector (gaussian pulse)
	#endif
	
	
	// Define input signal vector (NOISE)
	complex_t *As = (complex_t*)malloc(nBytes);
	NoiseGeneratorCPU ( As, SIZE );
	
	#ifdef THREE_EQS	
	// Define input idler vector (NOISE)
	complex_t *Ai = (complex_t*)malloc(nBytes);
	NoiseGeneratorCPU ( Ai, SIZE );
	#endif
	
	
	// Intracavy phase modulator
	bool using_phase_modulator = atoi(argv[9]);
	real_t mod_depth, fpm, df;
	if(using_phase_modulator){
		mod_depth       = atof(argv[10])*PI;
		df              = atof(argv[11])*sqrtf(Nth-1)*alphas/(PI*mod_depth)*FSR;
		fpm             = FSR - df;
	}
	
	
	// Define string variables for saving files
	std::string Filename, SAux, Extension = ".dat";
	bool save_input_fields = false;  // Save input fields files
	if (save_input_fields){
		#ifdef CW_OPO
		Filename = "pump_input";	SaveVectorComplexGPU (Ap_in, SIZE, Filename);
		#endif
		#ifdef NS_OPO
		Filename = "pump_input";	SaveVectorComplexGPU (Ap_in, SIZEL, Filename);
		#endif
		Filename = "signal_input";	SaveVectorComplexGPU (As, SIZE, Filename);
		#ifdef THREE_EQS	
		Filename = "idler_input";	SaveVectorComplexGPU (Ai, SIZE, Filename);	
		#endif
	}
	
	
	bool print_param_on_screen = true;	// Print parameters on screen
	if ( print_param_on_screen ){
		std::cout << "\n\nSimulation parameters:\n\n " << std::endl;
		std::cout << "Number of round trips   = " << NRT  << std::endl;
		std::cout << "Pump wavelength         = " << lp*1e3 << " nm" << std::endl;
		std::cout << "Signal wavelength       = " << ls*1e3 << " nm" << std::endl;
		#ifdef THREE_EQS
		std::cout << "Idler wavelength        = " << li*1e3 << " nm" << std::endl;
		#endif
		std::cout << "Temp                    = " << Temp << " ºC" << std::endl;
		std::cout << "np                      = " << np << std::endl;
		std::cout << "ns                      = " << ns << std::endl;
		std::cout << "ni                      = " << ni << std::endl;
		std::cout << "\u03BD⁻¹ pump                = " << 1.0/vp << " ps/\u03BCm" << std::endl;
		std::cout << "\u03BD⁻¹ signal              = " << 1.0/vs << " ps/\u03BCm" << std::endl;
		#ifdef THREE_EQS
		std::cout << "\u03BD⁻¹ idler               = " << 1.0/vi << " ps/\u03BCm" << std::endl;		
		#endif
		std::cout << "\u0394k                      = " << dk << " \u03BCm⁻¹" << std::endl;
		std::cout << "\u0394k'                     = " << dkp << " ps/\u03BCm" << std::endl;	
		std::cout << "GVD pump                = " << b2p << " ps²/\u03BCm" << std::endl;
		std::cout << "GVD signal              = " << b2s << " ps²/\u03BCm" << std::endl;
		#ifdef THREE_EQS
		std::cout << "GVD idler               = " << b2i << " ps²/\u03BCm" << std::endl;		
		#endif
		std::cout << "TOD pump                = " << b3p << " ps³/\u03BCm" << std::endl;
		std::cout << "TOD signal              = " << b3s << " ps³/\u03BCm" << std::endl;		
		std::cout << "Net GVD                 = " << (1-epsilon)*b2s << " ps²/\u03BCm" << std::endl;
		std::cout << "GVD compensation        = " << atoi(argv[7]) << " %"  << std::endl;
		std::cout << "Net TOD                 = " << (1-0.01*atoi(argv[8]))*b3s*Lcr*1e3 << " fs³"  << std::endl;
		std::cout << "TOD compensation        = " << atof(argv[8]) << " %"  << std::endl;		
		std::cout << "deff                    = " << deff*1e6 << " pm/V"  << std::endl;
		std::cout << "\u039B                       = " << Lambda << " \u03BCm"  << std::endl;
		std::cout << "\u03B1cp                     = " << alpha_crp << " \u03BCm⁻¹"  << std::endl;
		std::cout << "\u03B1cs                     = " << alpha_crs << " \u03BCm⁻¹" << std::endl;
		std::cout << "\u03B1s                      = " << alphas << std::endl;
		#ifdef THREE_EQS
		std::cout << "\u03B1ci                      = " << alpha_cri << " \u03BCm⁻¹" << std::endl;
		std::cout << "\u03B1i                      = " << alphas << std::endl;
		#endif
		
		std::cout << "Crystal length          = " << Lcr*1e-3 << " mm"  << std::endl;
		std::cout << "Cavity  length          = " << Lcav*1e-3 << " mm"  << std::endl;
		std::cout << "\u0394z                      = " << dz << " \u03BCm"  << std::endl;
		std::cout << "Reflectivity (signal)   = " << Rs*100 << " %"  << std::endl;
		#ifdef THREE_EQS
		std::cout << "Reflectivity (idler)    = " << Ri*100 << " %"  << std::endl;	
		#endif
		std::cout << "Cavity Finesse          = " << finesse << std::endl;	
		std::cout << "Cavity lw (FWHM)        = " << lw << " MHz"  << std::endl;	
		std::cout << "Round-trip time         = " << std::setprecision(15) << t_rt << " ps"  << std::endl;	
		std::cout << "FSR                     = " << std::setprecision(15) << FSR*1e3 << " GHz"  << std::endl;
		std::cout << "Cavity detuning (\u03B4)     = " << delta << "\u03C0"  << std::endl;	
		std::cout << "Using N                 = " << SIZE << " points" << std::endl;
		std::cout << "dT                      = " << dT << " ps" << std::endl;
		std::cout << "SIZEL                   = " << SIZEL << std::endl;
		std::cout << "dFp                     = " << dFp << " THz" << std::endl;
		std::cout << "Max frequency           = " << Fp[SIZEL-1] << " THz" << std::endl;
		std::cout << "Ap0                     = " << Ap0 << " V/um" << std::endl; 
		std::cout << "waist                   = " << waist << " \u03BCm" << std::endl;
		std::cout << "spot                    = " << spot << " \u03BCm²" << std::endl;
		std::cout << "Power threshold         = " << Pth << " W" << std::endl;
		std::cout << "Power                   = " << Power << " W" << std::endl;
		std::cout << "Times above the thres.  = " << Nth << std::endl;
		if(using_phase_modulator){
			std::cout << "Using a phase modulator:" << std::endl;
			std::cout << "Mod. depth (\u03B2)          = " << atof(argv[10]) << "\u03C0 rad = " << mod_depth << " rad" << std::endl;
			std::cout << "Freq. detuning (\u03B4f)     = " << df*1e6 << " MHz" << std::endl;
			std::cout << "Mod. frequency(fm)      = " << fpm*1e3 << " GHz" << std::endl;
			std::cout << "\n\nPoint in the space of parameters:\n" << std::endl;
			std::cout << "(N,\u03B2,\u03B4f,\u03B5) = ( " << Nth << ", " << atof(argv[10]) << ", "  << std::setprecision(4) << df*1e6 << ", " << epsilon << " )\n\n" << std::endl;			
		}
		else{std::cout << "No phase modulator" << std::endl;
			std::cout << "\n\nPoint in the space of parameters:\n" << std::endl;
			std::cout << "( N, \u03B2, \u03B4f, \u03B5 ) = ( " << Nth << ", 0, 0, " << std::setprecision(2) << epsilon << " )\n\n" << std::endl;
		}
	}
	////////////////////////////////////////////////////////////////////////////////////////
	
	
	
	
	////////////////////////////////////////////////////////////////////////////////////////
	//* Define GPU vectors */
	
	// Parameters for kernels
	dim3 block(BLKX);
	dim3 grid((SIZE+BLKX-1)/BLKX);
	
	
	// Define GPU vectors //
	real_t *w_gpu; // angular frequency 
	CHECK(hipMalloc((void **)&w_gpu, sizeof(real_t) * SIZE ));
	
	real_t *T_gpu; // time for a single round trip
	CHECK(hipMalloc((void **)&T_gpu, sizeof(real_t) * SIZE ));    
	CHECK(hipMemcpy(T_gpu, T, sizeof(real_t)*SIZE, hipMemcpyHostToDevice));    
	
	
	complex_t *Ap_gpu, *Ap_in_gpu, *Ap_total_gpu, *Apw_gpu, *As_gpu, *As_total_gpu, *Asw_gpu;
	CHECK(hipMalloc((void **)&As_gpu, nBytes ));
	CHECK(hipMalloc((void **)&Ap_gpu, nBytes ));
	
	#ifdef CW_OPO
	CHECK(hipMalloc((void **)&Ap_in_gpu, nBytes ));
	#endif
	#ifdef NS_OPO
	CHECK(hipMalloc((void **)&Ap_in_gpu, sizeof(complex_t)*SIZEL ));
	#endif
	CHECK(hipMalloc((void **)&As_total_gpu, sizeof(complex_t) * SIZEL ));
	CHECK(hipMalloc((void **)&Ap_total_gpu, sizeof(complex_t) * SIZEL ));
	CHECK(hipMalloc((void **)&Asw_gpu, nBytes ));
	CHECK(hipMalloc((void **)&Apw_gpu, nBytes ));
	
	CHECK(hipMemcpy(As_gpu, As, nBytes, hipMemcpyHostToDevice));
	#ifdef CW_OPO
	CHECK(hipMemcpy(Ap_in_gpu, Ap_in, nBytes, hipMemcpyHostToDevice));	
	#endif
	#ifdef NS_OPO
	CHECK(hipMemcpy(Ap_in_gpu, Ap_in, sizeof(complex_t) * SIZEL, hipMemcpyHostToDevice));
	#endif
	
	CHECK(hipMemcpy(w_gpu, w, sizeof(real_t) * SIZE , hipMemcpyHostToDevice));
	
	
	// RK4 (kx) and auxiliary (aux) GPU vectors 
	complex_t *k1p_gpu, *k2p_gpu, *k3p_gpu, *k4p_gpu, *k1s_gpu, *k2s_gpu, *k3s_gpu, *k4s_gpu;
	CHECK(hipMalloc((void **)&k1p_gpu, nBytes ));	CHECK(hipMalloc((void **)&k2p_gpu, nBytes ));
	CHECK(hipMalloc((void **)&k3p_gpu, nBytes ));	CHECK(hipMalloc((void **)&k4p_gpu, nBytes ));
	CHECK(hipMalloc((void **)&k1s_gpu, nBytes ));	CHECK(hipMalloc((void **)&k2s_gpu, nBytes ));
	CHECK(hipMalloc((void **)&k3s_gpu, nBytes ));	CHECK(hipMalloc((void **)&k4s_gpu, nBytes ));
	
	complex_t *auxp_gpu, *auxs_gpu;
	CHECK(hipMalloc((void **)&auxp_gpu, nBytes ));	CHECK(hipMalloc((void **)&auxs_gpu, nBytes ));
	
	#ifdef THREE_EQS	
	complex_t *Ai_gpu, *Ai_total_gpu, *Aiw_gpu;
	CHECK(hipMalloc((void **)&Ai_gpu, nBytes ));
	CHECK(hipMalloc((void **)&Ai_total_gpu, sizeof(complex_t) * SIZEL ));
	CHECK(hipMalloc((void **)&Aiw_gpu, nBytes ));
	
	CHECK(hipMemcpy(Ai_gpu, Ai, nBytes, hipMemcpyHostToDevice));	
	
	complex_t *k1i_gpu, *k2i_gpu, *k3i_gpu, *k4i_gpu, *auxi_gpu;
	CHECK(hipMalloc((void **)&k1i_gpu, nBytes ));	CHECK(hipMalloc((void **)&k2i_gpu, nBytes ));
	CHECK(hipMalloc((void **)&k3i_gpu, nBytes ));	CHECK(hipMalloc((void **)&k4i_gpu, nBytes ));
	CHECK(hipMalloc((void **)&auxi_gpu, nBytes ));
	
	bool idler_pm = true; // phase modulator applies on idler
	#endif
	
	////////////////////////////////////////////////////////////////////////////////////////
	
	
	
	
	////////////////////////////////////////////////////////////////////////////////////////	
	//* Main loop. Fields in the cavity */
	
	// Set plan for cuFFT //
	hipfftHandle plan1D; 
	hipfftPlan1d(&plan1D, SIZE, HIPFFT_C2C, 1);
	
	std::cout << "Starting main loop on CPU & GPU...\n" << std::endl;
	uint mm = 0; // counts for cw saved round trips
	for (uint nn = 0; nn < NRT; nn++){
		if( nn%250 == 0 or nn == NRT-1 )
			std::cout << "#round trip: " << nn << std::endl;
		
		#ifdef CW_OPO
		// update the input pump in each round trip
		CHECK(hipMemcpy( Ap_gpu, Ap_in_gpu, nBytes, hipMemcpyDeviceToDevice) );
		#endif
		#ifdef NS_OPO
		// read the input pump in nn-th round trip
		ReadPump<<<grid,block>>>( Ap_gpu, Ap_in_gpu, NRT, nn );
		CHECK(hipDeviceSynchronize()); 
		#endif
		
		#ifdef THREE_EQS
		if (!is_Ai_resonant){	// For non-resonant field, it resets Ai in every round trip
			NoiseGeneratorCPU ( Ai, SIZE );
			CHECK(hipMemcpy(Ai_gpu, Ai, nBytes, hipMemcpyHostToDevice));
		}
		#endif
		
		if (!is_As_resonant){	// For non-resonant field, it resets As in every round trip
			NoiseGeneratorCPU ( As, SIZE );
			CHECK(hipMemcpy(As_gpu, As, nBytes, hipMemcpyHostToDevice));
		}
		
		#ifdef THREE_EQS // Single pass for coupled wave equations (2 or 3)
		EvolutionInCrystal( w_gpu, Ap_gpu, As_gpu, Ai_gpu, Apw_gpu, Asw_gpu, Aiw_gpu, k1p_gpu, k1s_gpu, k1i_gpu, k2p_gpu, k2s_gpu, k2i_gpu, k3p_gpu, k3s_gpu, k3i_gpu, k4p_gpu, k4s_gpu, k4i_gpu, auxp_gpu, auxs_gpu, auxi_gpu, vp, vs, vi, b2p, b2s, b2i, b3p, b3s, b3i, dk, kp, ks, ki, dz );
		#else
		EvolutionInCrystal( w_gpu, Ap_gpu, As_gpu, Apw_gpu, Asw_gpu, k1p_gpu, k1s_gpu, k2p_gpu, k2s_gpu, k3p_gpu, k3s_gpu, k4p_gpu, k4s_gpu, auxp_gpu, auxs_gpu, vp, vs, b2p, b2s, b3p, b3s, dk, kp, ks, dz );
		#endif
		
		
		if(GDD!=0){ // adds dispersion compensation
			hipfftExecC2C(plan1D, (complex_t *)As, (complex_t *)Asw_gpu, HIPFFT_BACKWARD);
			CHECK(hipDeviceSynchronize());
			CUFFTscale<<<grid,block>>>(Asw_gpu, SIZE);
			CHECK(hipDeviceSynchronize());
			AddGDD<<<grid,block>>>(Asw_gpu, auxs_gpu, w_gpu, GDD);
			CHECK(hipDeviceSynchronize());
			hipfftExecC2C(plan1D, (complex_t *)Asw_gpu, (complex_t *)As_gpu, HIPFFT_FORWARD);
			CHECK(hipDeviceSynchronize());
			#ifdef THREE_EQS
			hipfftExecC2C(plan1D, (complex_t *)Ai, (complex_t *)Aiw_gpu, HIPFFT_BACKWARD);
			CHECK(hipDeviceSynchronize());
			CUFFTscale<<<grid,block>>>(Aiw_gpu, SIZE);
			CHECK(hipDeviceSynchronize());
			AddGDD<<<grid,block>>>(Aiw_gpu, auxi_gpu, w_gpu, GDD);
			CHECK(hipDeviceSynchronize());
			hipfftExecC2C(plan1D, (complex_t *)Aiw_gpu, (complex_t *)Ai_gpu, HIPFFT_FORWARD);
			CHECK(hipDeviceSynchronize());
			#endif
		}			
		
		if( using_phase_modulator ){ // use an intracavy phase modulator of one o more fields
			PhaseModulatorIntraCavity<<<grid,block>>>(As_gpu, auxs_gpu, mod_depth, fpm, T_gpu);
			CHECK(hipDeviceSynchronize());
			#ifdef THREE_EQS
			if(idler_pm){
				PhaseModulatorIntraCavity<<<grid,block>>>(Ai_gpu, auxi_gpu, mod_depth, fpm, T_gpu);
				CHECK(hipDeviceSynchronize());
			}
			#endif
		}
		
		if (is_As_resonant){ // if As is resonant, adds phase and losses
			AddPhase<<<grid,block>>>(As_gpu, auxs_gpu, Rs, delta, nn);
			CHECK(hipDeviceSynchronize());
		}
		
		#ifdef THREE_EQS
		if (is_Ai_resonant){  // if Ai is resonant, adds phase and losses
			AddPhase<<<grid,block>>>(Ai_gpu, auxi_gpu, Ri, delta, nn);
			CHECK(hipDeviceSynchronize());
		}
		#endif
		
		#ifdef CW_OPO	// saves systematically every round trip
		if (stride){  
			if (nn % 100 == 0){ // this branch is useful if the user want to save the round trips every 100 ones
				std::cout << "Saving the " << nn << "-th round trip" << std::endl;
				SaveRoundTrip<<<grid,block>>>(As_total_gpu, As_gpu, mm, Nrts ); // saves signal
				CHECK(hipDeviceSynchronize());
				SaveRoundTrip<<<grid,block>>>(Ap_total_gpu, Ap_gpu, mm, Nrts ); // saves pump
				CHECK(hipDeviceSynchronize());
				#ifdef THREE_EQS
				SaveRoundTrip<<<grid,block>>>(Ai_total_gpu, Ai_gpu, mm, Nrts ); // saves idler
				CHECK(hipDeviceSynchronize());
				#endif
				mm += 1;
			}			
		}
		else{  // this branch is useful if the user want to save the last NRT-Nrts round trips
			if (nn >= NRT -Nrts){                
				SaveRoundTrip<<<grid,block>>>( As_total_gpu, As_gpu, mm, Nrts ); // saves signal
				CHECK(hipDeviceSynchronize());
				SaveRoundTrip<<<grid,block>>>( Ap_total_gpu, Ap_gpu, mm, Nrts ); // saves pump
				CHECK(hipDeviceSynchronize());
				#ifdef THREE_EQS
				SaveRoundTrip<<<grid,block>>>( Ai_total_gpu, Ai_gpu, mm, Nrts ); // saves idler
				CHECK(hipDeviceSynchronize());
				#endif
				mm += 1;
			}
		}
		#endif
		#ifdef NS_OPO	// save the simulation in the NS regime
		SaveRoundTrip<<<grid,block>>>(Ap_total_gpu, Ap_gpu, nn, NRT ); // saves signal
		CHECK(hipDeviceSynchronize());
		SaveRoundTrip<<<grid,block>>>(As_total_gpu, As_gpu, nn, NRT ); // saves pump
		CHECK(hipDeviceSynchronize());
		#ifdef THREE_EQS
		SaveRoundTrip<<<grid,block>>>(Ai_total_gpu, Ai_gpu, nn, NRT ); // saves idler
		CHECK(hipDeviceSynchronize());
		#endif
		#endif
		
	} // End of main loop
	
	////////////////////////////////////////////////////////////////////////////////////////
	
	
	
	
	////////////////////////////////////////////////////////////////////////////////////////	
	//* Saving results in .dat files using the function SaveVectorComplexGPU() //
	
	bool save_vectors = true; // Decide whether or not save the following vectors
	if (save_vectors){
		std::cout << "\nSaving time and frequency vectors...\n" << std::endl;
		Filename = "Tp"; SaveVectorReal (Tp, SIZEL, Filename+Extension);
		Filename = "freq"; SaveVectorReal (Fp, SIZEL, Filename+Extension);
		Filename = "T"; SaveVectorReal (T, SIZE, Filename+Extension);
	}
	else{ std::cout << "\nTime and frequency were previuosly save...\n" << std::endl;
	}
	
	// Save the full simulation
	Filename = "signal_output";	SaveVectorComplexGPU ( As_total_gpu, SIZEL, Filename );
	Filename = "pump_output";	SaveVectorComplexGPU ( Ap_total_gpu, SIZEL, Filename );
	#ifdef THREE_EQS
	Filename = "idler_output";	SaveVectorComplexGPU ( Ai_total_gpu, SIZEL, Filename );
	#endif
	
	////////////////////////////////////////////////////////////////////////////////////////
	
	
	
	
	////////////////////////////////////////////////////////////////////////////////////////
	//* Deallocating memory from CPU and GPU and destroying plans */
	
	free(Tp); free(T); free(Fp); free(w); free(F);
	free(As); free(Ap_in);
	
	CHECK(hipFree(As_gpu)); 		CHECK(hipFree(Ap_gpu));
	CHECK(hipFree(As_total_gpu));	CHECK(hipFree(Ap_total_gpu));
	CHECK(hipFree(Ap_in_gpu));	
	CHECK(hipFree(T_gpu)); 		CHECK(hipFree(w_gpu));
	CHECK(hipFree(k1p_gpu));		CHECK(hipFree(k2p_gpu));
	CHECK(hipFree(k3p_gpu));        	CHECK(hipFree(k4p_gpu));
	CHECK(hipFree(k1s_gpu));        	CHECK(hipFree(k2s_gpu));
	CHECK(hipFree(k3s_gpu));        	CHECK(hipFree(k4s_gpu));	
	CHECK(hipFree(auxs_gpu));       	CHECK(hipFree(auxp_gpu));
	
	#ifdef THREE_EQS
	free(Ai); 
	
	CHECK(hipFree(Ai_gpu));	CHECK(hipFree(Ai_total_gpu));
	CHECK(hipFree(k1i_gpu));     CHECK(hipFree(k2i_gpu));
	CHECK(hipFree(k3i_gpu));     CHECK(hipFree(k4i_gpu));
	CHECK(hipFree(auxi_gpu));
	#endif
	
	// Destroy CUFFT context and reset the GPU
	hipfftDestroy(plan1D); 	hipDeviceReset();    
	
	////////////////////////////////////////////////////////////////////////////////////////
	
	
	
	
	////////////////////////////////////////////////////////////////////////////////////////	
	//* Finish timing: returns the runtime simulation */
	
	double iElaps = seconds() - iStart;
	if(iElaps>60){std::cout << "\n\n...time elapsed " <<  iElaps/60.0 << " min\n\n " << std::endl;}
	else{std::cout << "\n\n...time elapsed " <<  iElaps << " seconds\n\n " << std::endl;}
	
	time(&current_time);
	std::cout << ctime(&current_time) << std::endl;
	
	////////////////////////////////////////////////////////////////////////////////////////
	
	return 0;
}
